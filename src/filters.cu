#include "hip/hip_runtime.h"
#include "include/filters.h"
#include <hip/hip_runtime.h>
#include <>
#include <opencv2/opencv.hpp>
#include <vector>
#include <string>
#include <filesystem> 

#define CHECK_CUDA_CALL(call)                                                     \
    {                                                                            \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            throw std::runtime_error(std::string("CUDA Error: ") +               \
                                     hipGetErrorString(err) +                   \
                                     " at " + __FILE__ + ":" + std::to_string(__LINE__)); \
        }                                                                        \
    }

// CUDA Kernel
__global__ void applyFilterKernel(const float* image, const float* filter, float* output, int width, int height, int filterWidth) {
    extern __shared__ float sharedMemory[]; // Shared memory buffer
    float* sharedImage = sharedMemory;      // Shared image tile
    float* sharedFilter = &sharedMemory[(blockDim.x + 2) * (blockDim.y + 2)]; // Shared filter
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int halfFilter = filterWidth / 2;

    // Load the filter into shared memory
    if (tx < filterWidth && ty < filterWidth) {
        sharedFilter[ty * filterWidth + tx] = (tx < filterWidth && ty < filterWidth) ? filter[ty * filterWidth + tx] : 0.0f;
    }

    // Load the image tile into shared memory (with padding for boundary handling)
    int sharedWidth = blockDim.x + 2 * halfFilter;
    int sharedHeight = blockDim.y + 2 * halfFilter;

    int sharedX = tx + halfFilter;
    int sharedY = ty + halfFilter;

    if (x < width && y < height) {
        sharedImage[sharedY * sharedWidth + sharedX] = image[y * width + x];
    } else {
        sharedImage[sharedY * sharedWidth + sharedX] = 0.0f;
    }

    // Handle boundary padding
    if (tx < halfFilter) {
        sharedImage[sharedY * sharedWidth + (sharedX - halfFilter)] = (x >= halfFilter) ? image[y * width + (x - halfFilter)] : 0.0f;
        sharedImage[sharedY * sharedWidth + (sharedX + blockDim.x)] = (x + blockDim.x < width) ? image[y * width + (x + blockDim.x)] : 0.0f;
    }
    if (ty < halfFilter) {
        sharedImage[(sharedY - halfFilter) * sharedWidth + sharedX] = (y >= halfFilter) ? image[(y - halfFilter) * width + x] : 0.0f;
        sharedImage[(sharedY + blockDim.y) * sharedWidth + sharedX] = (y + blockDim.y < height) ? image[(y + blockDim.y) * width + x] : 0.0f;
    }

    __syncthreads();

    // Perform convolution
    if (x < width && y < height) {
        float value = 0.0f;
        for (int fy = -halfFilter; fy <= halfFilter; ++fy) {
            for (int fx = -halfFilter; fx <= halfFilter; ++fx) {
                value += sharedImage[(sharedY + fy) * sharedWidth + (sharedX + fx)] *
                         sharedFilter[(fy + halfFilter) * filterWidth + (fx + halfFilter)];
            }
        }
        output[y * width + x] = value;
    }
}

void applyFiltersCUDA(const cv::Mat& channel, const std::vector<cv::Mat>& filterBank, std::vector<cv::Mat>& responses) {
    if (channel.type() != CV_32F) {
        throw std::invalid_argument("Input channel must have type CV_32F");
    }

    int width = channel.cols;
    int height = channel.rows;

    // Allocate device memory
    float* d_image;
    CHECK_CUDA_CALL(hipMalloc((void**)&d_image, width * height * sizeof(float)));
    CHECK_CUDA_CALL(hipMemcpy(d_image, channel.ptr<float>(), width * height * sizeof(float), hipMemcpyHostToDevice));

    float* d_filter;
    float* d_output;
    int maxFilterSize = 0;
    for (const auto& filter : filterBank) {
        maxFilterSize = std::max(maxFilterSize, filter.rows);
    }
    CHECK_CUDA_CALL(hipMalloc((void**)&d_filter, maxFilterSize * maxFilterSize * sizeof(float)));
    CHECK_CUDA_CALL(hipMalloc((void**)&d_output, width * height * sizeof(float)));

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    for (const auto& filter : filterBank) {
        int filterWidth = filter.cols;
        cv::Mat filter32F;
        if (filter.type() != CV_32F) {
            filter.convertTo(filter32F, CV_32F);
        } else {
            filter32F = filter;
        }

        CHECK_CUDA_CALL(hipMemcpy(d_filter, filter32F.ptr<float>(), filterWidth * filterWidth * sizeof(float), hipMemcpyHostToDevice));

        size_t sharedMemSize = ((blockSize.x + 2) * (blockSize.y + 2) + filterWidth * filterWidth) * sizeof(float);
        applyFilterKernel<<<gridSize, blockSize, sharedMemSize>>>(d_image, d_filter, d_output, width, height, filterWidth);
        CHECK_CUDA_CALL(hipDeviceSynchronize());

        cv::Mat response(height, width, CV_32F);
        CHECK_CUDA_CALL(hipMemcpy(response.ptr<float>(), d_output, width * height * sizeof(float), hipMemcpyDeviceToHost));
        responses.push_back(response);
    }

    // Free device memory
    hipFree(d_image);
    hipFree(d_filter);
    hipFree(d_output);
}

void extractFilterResponsesCUDA(const cv::Mat& image, const std::vector<cv::Mat>& filterBank, const std::string& outputPath) {
    if (image.empty()) {
        throw std::invalid_argument("Input image is empty!");
    }
    std::filesystem::create_directories(outputPath);

    cv::Mat labImage;
    cv::cvtColor(image, labImage, cv::COLOR_BGR2Lab);

    std::vector<cv::Mat> labChannels(3);
    cv::split(labImage, labChannels);

    for (int i = 0; i < 3; ++i) {
        labChannels[i].convertTo(labChannels[i], CV_32F);
    }

    std::vector<std::vector<cv::Mat>> channelResponses(3);

    for (int i = 0; i < 3; ++i) {
        applyFiltersCUDA(labChannels[i], filterBank, channelResponses[i]);
    }

    for (int i = 0; i < filterBank.size(); ++i) {
        saveFilterResponseImage(channelResponses[0][i], outputPath, i + 1, "L");
        saveFilterResponseImage(channelResponses[1][i], outputPath, i + 1, "a");
        saveFilterResponseImage(channelResponses[2][i], outputPath, i + 1, "b");
    }
}
